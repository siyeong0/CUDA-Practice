#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <random>

__global__ void addKernel(const int* a, const int* b, int* c, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
	{
		c[i] = a[i] + b[i];
	}
}

int main()
{
	const int threadsPerBlock = 256;
	const int size = 1024 * 1024 * 256;

	int* a = nullptr;
	int* b = nullptr;
	int* c = nullptr;
	int* cc = nullptr;

	// pinned-memory
	hipHostMalloc(&a, sizeof(int) * size);
	hipHostMalloc(&b, sizeof(int) * size);
	hipHostMalloc(&c, sizeof(int) * size);
	hipHostMalloc(&cc, sizeof(int) * size);

	for (int i = 0; i < size; i++)
	{
		a[i] = std::rand() % 10;
		b[i] = rand() % 10;
		cc[i] = a[i] + b[i];
	}

	// Add vectors using stream.
	{
		hipStream_t stream;
		hipStreamCreate(&stream);

		int* dev_a = nullptr;
		int* dev_b = nullptr;
		int* dev_c = nullptr;

		hipMalloc((void**)&dev_a, size * sizeof(int)); // input a
		hipMalloc((void**)&dev_b, size * sizeof(int)); // input b
		hipMalloc((void**)&dev_c, size * sizeof(int)); // output c

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start, 0);

		hipMemcpyAsync(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice, stream); // 비동기적으로 복사 복사
		hipMemcpyAsync(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice, stream);

		int numBlocks = int(ceil(float(size) / threadsPerBlock)); // 블럭 여러 개 사용
		addKernel << <numBlocks, threadsPerBlock, 0, stream >> > (dev_a, dev_b, dev_c, size);

		hipMemcpyAsync(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost, stream);

		hipEventRecord(stop, 0);  // 끝나는 시간 기록
		hipDeviceSynchronize();   // kernel이 끝날때까지 대기 (동기화)

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop); // 걸린 시간 계산
		std::cout << "Time elapsed: " << milliseconds << " ms" << std::endl;

		for (int i = 0; i < size; i++)
		{
			if (cc[i] - c[i] > 1e-4)
			{
				std::cout << "Wrong result" << std::endl;
				goto EXIT;
			}
		}

		std::cout << "Correct" << std::endl;

	EXIT:
		hipEventDestroy(start);
		hipEventDestroy(stop);

		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);

		hipHostFree(a);
		hipHostFree(b);
		hipHostFree(cc);
		hipHostFree(c);

		hipDeviceReset();
	}

	return 0;
}

