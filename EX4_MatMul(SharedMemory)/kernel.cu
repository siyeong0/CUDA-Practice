#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <iomanip>
#include <vector>

constexpr int BLOCK_SIZE = 32;

struct Matrix
{
	int Height = 0;
	int Width = 0;
	int Stride = 0;
	float* Data = nullptr;
};

void matMulCPU(const Matrix& a, const Matrix& b, Matrix* out)
{
	int M = a.Height;
	int K = a.Width;
	int N = b.Width;
	for (int row = 0; row < M; ++row)
	{
		for (int col = 0; col < N; ++col) {
			float v = 0.0f;
			for (int e = 0; e < K; e++)
			{
				v += a.Data[row * a.Width + e] * b.Data[e * b.Width + col];
			}
			out->Data[row * out->Width + col] = v;
		}
	}
}

__device__ Matrix getSubMatrix(Matrix mat, int row, int col) {
	Matrix sub;
	sub.Width = BLOCK_SIZE;
	sub.Height = BLOCK_SIZE;
	sub.Stride = mat.Stride;
	sub.Data = &mat.Data[mat.Stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
	return sub;
}

__global__ void matMulKernel(Matrix matA, Matrix matB, Matrix matOut)
{
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;

	Matrix subMatOut = getSubMatrix(matOut, blockRow, blockCol);

	int row = threadIdx.y;
	int col = threadIdx.x;

	float v = 0.0f;
	for (int i = 0; i < matA.Width / BLOCK_SIZE; ++i)
	{
		Matrix subMatA = getSubMatrix(matA, blockRow, i);
		Matrix subMatB = getSubMatrix(matB, i, blockCol);

		__shared__ float sharedMatA[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float sharedMatB[BLOCK_SIZE][BLOCK_SIZE];

		sharedMatA[row][col] = subMatA.Data[row * subMatA.Stride + col];
		sharedMatB[row][col] = subMatB.Data[row * subMatB.Stride + col];

		__syncthreads();

		for (int j = 0; j < BLOCK_SIZE; ++j)
		{
			v += sharedMatA[row][j] * sharedMatB[j][col];
		}

		__syncthreads();
	}

	subMatOut.Data[row * subMatOut.Stride + col] = v;
}

int main()
{
	const int M = 1024 * 2;
	const int N = 1024 * 1;
	const int K = 256;

	Matrix matA{ M, K };
	Matrix matB{ K, N };
	Matrix matC{ M, N };
	Matrix matCC{ M, N };

	matA.Stride = matA.Width;
	matB.Stride = matB.Width;
	matC.Stride = matC.Width;
	matCC.Stride = matCC.Width;

	matA.Data = new float[matA.Width * matA.Height];
	for (int i = 0; i < matA.Width * matA.Height; i++) matA.Data[i] = 0.1f * (float)(std::rand() % 10);
	matB.Data = new float[matB.Width * matB.Height];
	for (int i = 0; i < matB.Width * matB.Height; i++) matB.Data[i] = 0.1f * (float)(std::rand() % 10);
	matC.Data = new float[matC.Width * matC.Height];
	for (int i = 0; i < matC.Width * matC.Height; i++) matC.Data[i] = 0.0f;

	matCC.Data = new float[matCC.Width * matCC.Height];
	matMulCPU(matA, matB, &matCC);

	// Sum vectors in parallel.
	{
		Matrix devMatA{ M,K };
		Matrix devMatB{ K,N };
		Matrix devMatC{ M,N };
		devMatA.Stride = devMatA.Width;
		devMatB.Stride = devMatB.Width;
		devMatC.Stride = devMatC.Width;

		hipError_t cudaStatus;

		cudaStatus = hipSetDevice(0);

		cudaStatus = hipMalloc((void**)&devMatA.Data, devMatA.Width * devMatA.Height * sizeof(float));
		cudaStatus = hipMalloc((void**)&devMatB.Data, devMatB.Width * devMatB.Height * sizeof(float));
		cudaStatus = hipMalloc((void**)&devMatC.Data, devMatC.Width * devMatC.Height * sizeof(float));

		cudaStatus = hipMemcpy(devMatA.Data, matA.Data, devMatA.Width * devMatA.Height * sizeof(float), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(devMatB.Data, matB.Data, devMatB.Width * devMatB.Height * sizeof(float), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(devMatC.Data, matC.Data, devMatC.Width * devMatC.Height * sizeof(float), hipMemcpyHostToDevice);

		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
		dim3 dimGrid(devMatC.Width / dimBlock.x, devMatC.Height / dimBlock.y); // Assert no remainder

		matMulKernel << <dimGrid, dimBlock >> > (devMatA, devMatB, devMatC);

		cudaStatus = hipDeviceSynchronize();

		cudaStatus = hipMemcpy(matC.Data, devMatC.Data, devMatC.Width * devMatC.Height * sizeof(float), hipMemcpyDeviceToHost);

		hipFree(devMatA.Data);
		hipFree(devMatB.Data);
		hipFree(devMatC.Data);

		cudaStatus = hipDeviceReset();
	}

	// Check result.
	bool bCorrect = true;
	for (int i = 0; i < matC.Width * matC.Height; i++)
	{
		bCorrect &= (1e-4 > std::fabsf(matC.Data[i] - matCC.Data[i]));
	}

	std::string resultPhrase = bCorrect ? "Correct!" : "Wrong!";
	std::cout << resultPhrase << std::endl;

	// Free memory.
	delete[] matA.Data;
	delete[] matB.Data;
	delete[] matC.Data;
	delete[] matCC.Data;

	return 0;
}