#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>
#include <random>

// Error check macro.
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d in %s: %s\n", \
                    __FILE__, __LINE__, __func__, hipGetErrorString(err)); \
            exit(1); \
        } \
    } while (0)

__global__ void addKernel(const int* a, const int* b, int* c, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
	{
		c[i] = a[i] + b[i];
	}
}

int main()
{
	const int threadsPerBlock = 1024;
	const int size = 1024 * 1024 * 512;
	const int numSplits = 8;
	const int splitSize = size / numSplits;

	int* a = nullptr;
	int* b = nullptr;
	int* c = nullptr;

	hipHostMalloc(&a, sizeof(int) * size); // pinned-memory
	hipHostMalloc(&b, sizeof(int) * size);
	hipHostMalloc(&c, sizeof(int) * size);

	for (int i = 0; i < size; i++)
	{
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	// Add large vector.
	{
		hipStream_t stream;
		hipStreamCreate(&stream);

		int* dev_a = nullptr;
		int* dev_b = nullptr;
		int* dev_c = nullptr;

		CUDA_CHECK(hipMalloc((void**)&dev_a, splitSize * sizeof(int)));
		CUDA_CHECK(hipMalloc((void**)&dev_b, splitSize * sizeof(int)));
		CUDA_CHECK(hipMalloc((void**)&dev_c, splitSize * sizeof(int)));

		hipEvent_t start, stop;
		CUDA_CHECK(hipEventCreate(&start));
		CUDA_CHECK(hipEventCreate(&stop));

		CUDA_CHECK(hipEventRecord(start, 0));

		for (int s = 0; s < numSplits; s++)
		{
			CUDA_CHECK(hipMemcpyAsync(dev_a, &a[s * splitSize], splitSize * sizeof(int), hipMemcpyHostToDevice, stream)); // size -> split_size
			CUDA_CHECK(hipMemcpyAsync(dev_b, &b[s * splitSize], splitSize * sizeof(int), hipMemcpyHostToDevice, stream)); // size -> split_size

			int numBlocks = int(ceil(float(splitSize) / threadsPerBlock));
			addKernel << <numBlocks, threadsPerBlock, 0, stream >> > (dev_a, dev_b, dev_c, splitSize);

			CUDA_CHECK(hipMemcpyAsync(&c[s * splitSize], dev_c, splitSize * sizeof(int), hipMemcpyDeviceToHost, stream));
		}

		CUDA_CHECK(hipEventRecord(stop, 0));
		CUDA_CHECK(hipDeviceSynchronize());

		float milliseconds = 0;
		CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
		std::cout << "Time elapsed: " << milliseconds << " ms" << std::endl;

		for (int i = 0; i < size; i++)
		{
			if (c[i] != a[i] + b[i])
			{
				std::cout << "Wrong result" << std::endl;
				goto EXIT;
			}
		}

		std::cout << "Correct" << std::endl;

	EXIT:
		CUDA_CHECK(hipEventDestroy(start));
		CUDA_CHECK(hipEventDestroy(stop));

		CUDA_CHECK(hipFree(dev_c));
		CUDA_CHECK(hipFree(dev_a));
		CUDA_CHECK(hipFree(dev_b));

		CUDA_CHECK(hipHostFree(a));
		CUDA_CHECK(hipHostFree(b));
		CUDA_CHECK(hipHostFree(c));

		CUDA_CHECK(hipDeviceReset());
	}


	return 0;
}