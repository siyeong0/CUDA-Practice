#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <iomanip>
#include <vector>

__global__ void addKernel(int* c, const int* a, const int* b, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size)
	{
		c[i] = a[i] + b[i];
	}
}

int main()
{
	constexpr int size = 1024 * 1024;
	std::vector<int> a(size);
	std::vector<int> b(size);
	std::vector<int> c(size);
	std::vector<int> cc(size); // A vector for verifying correctness of result.

	for (int i = 0; i < size; ++i)
	{
		a[i] = std::rand() % 100;
		b[i] = std::rand() % 100;
		cc[i] = a[i] + b[i];
	}

	// Add vectors in parallel.
	{
		// The code that checks cudaStatus is omitted for brevity.
		int* dev_a = nullptr;
		int* dev_b = nullptr;
		int* dev_c = nullptr;
		hipError_t cudaStatus;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);

		// Allocate GPU buffers for three vectors (two input, one output).
		cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
		cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
		cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_a, a.data(), size * sizeof(int), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_b, b.data(), size * sizeof(int), hipMemcpyHostToDevice);

		// Launch a kernel on the GPU with one thread for each element.
		int numThreadsPerBlock = 256;
		int numBlocks = int(std::ceil(float(size) / numThreadsPerBlock));
		addKernel << <numBlocks, numThreadsPerBlock >> > (dev_c, dev_a, dev_b, size);

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c.data(), dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
	}

	bool bCorrect = true;
	for (int i = 0; i < size; ++i)
	{
		bCorrect &= c[i] == cc[i];
	}

	std::string resultPhrase = bCorrect ? "Correct!" : "Wrong!";
	std::cout << resultPhrase << std::endl;

	return 0;
}